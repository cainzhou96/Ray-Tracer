#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];

    float nDotWo = dot(tri.normal, -ray.direction);
    if (nDotWo == 0.0f) return;

    float t = dot(tri.v1 - ray.origin, tri.normal) / dot(ray.direction, tri.normal);
    float3 P = ray.origin + t * ray.direction; // intersection in the object space

    if (t < 0.001) return;

    float3 tmp0 = tri.v3 - tri.v1;
    float3 tmp1 = tri.v2 - tri.v1;
    float3 tmp2 = P - tri.v1;
    float tmp0dot0 = dot(tmp0, tmp0);
    float tmp0dot1 = dot(tmp0, tmp1);
    float tmp0dot2 = dot(tmp0, tmp2);
    float tmp1dot1 = dot(tmp1, tmp1);
    float tmp1dot2 = dot(tmp1, tmp2);
    float denom = tmp0dot0 * tmp1dot1 - tmp0dot1 * tmp0dot1;

    float u = (tmp1dot1 * tmp0dot2 - tmp0dot1 * tmp1dot2) / denom;
    float v = (tmp0dot0 * tmp1dot2 - tmp0dot1 * tmp0dot2) / denom;

    if (0 > u || u > 1 || 0 > v || v > 1 || u + v > 1) return;

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib.intersection = P;
        attrib.wo = -ray.direction;
        attrib.normal = nDotWo > 0 ? tri.normal : -tri.normal;
        attrib.mv = tri.mv;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    result[0] = fminf(fminf(tri.v1.x, tri.v2.x), tri.v3.x);
    result[1] = fminf(fminf(tri.v1.y, tri.v2.y), tri.v3.y);
    result[2] = fminf(fminf(tri.v1.z, tri.v2.z), tri.v3.z);
    result[3] = fmaxf(fmaxf(tri.v1.x, tri.v2.x), tri.v3.x);
    result[4] = fmaxf(fmaxf(tri.v1.y, tri.v2.y), tri.v3.y);
    result[5] = fmaxf(fmaxf(tri.v1.z, tri.v2.z), tri.v3.z);
}