#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

#define IS_HEMISPEHRE 0
#define IS_COSINE 1
#define IS_BRDF 2

#define NEE_OFF 0
#define NEE_ON 1
#define NEE_MIS 2

#define BRDF_PHONG 0
#define BRDF_GGX 1

using namespace optix;

// Declare light buffers
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(int, lightSamples, , );
rtDeclareVariable(int, lightStratify, , );
rtDeclareVariable(int, nee, , );
rtDeclareVariable(int, importanceSampling, , );
rtDeclareVariable(int, russianRoulette, , );

rtDeclareVariable(int, brdf, , );
rtDeclareVariable(float, roughness, , );
rtDeclareVariable(float, gamma, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

float3 transformRay(float3 ray, float3 w, float epsilon); 
float3 getCosineSampleRay(float epsilon); 
float3 getHemisphereSampleRay(float epsilon); 
float3 getBRDFSampleRay(Attributes attrib, float epsilon); 

float3 getPhongBRDF(Attributes attrib, float3 wi); 
float3 getGGXBRDF(Attributes attrib, float3 wi); 
float3 getGGXThroughput(Attributes attrib, float3& wi);

float getCosinePDF(); 
float getHemispherePDF(); 
float getBRDFPDF(Attributes attrib, float3 wi); 

RT_PROGRAM void pathTracer() {
    MaterialValue mv = attrib.mv;
    Config cf = config[0];
	float3 wi;
	float3 throughput;

	if (brdf == BRDF_GGX) {
		throughput = getGGXThroughput(attrib, wi);
	}
	else {

		// ### SAMPLE ###
		if (importanceSampling == IS_HEMISPEHRE) {
			wi = getHemisphereSampleRay(cf.epsilon);
		}
		else if (importanceSampling == IS_COSINE) {
			wi = getCosineSampleRay(cf.epsilon);
		}
		else if (importanceSampling == IS_BRDF) {
			wi = getBRDFSampleRay(attrib, cf.epsilon);
		}

		// ### BRDF ###
		float3 f = getPhongBRDF(attrib, wi);

		// ### PDF ###
		float pdf;
		int N = 1;
		if (importanceSampling == IS_HEMISPEHRE) {
			pdf = getHemispherePDF();
			throughput = f * clamp(dot(attrib.normal, wi), 0.0f, 1.0f) / pdf / N;
		}
		else if (importanceSampling == IS_COSINE) {
			pdf = getCosinePDF();
			throughput = f / pdf / N;
		}
		else if (importanceSampling == IS_BRDF) {
			pdf = getBRDFPDF(attrib, wi);
			throughput = f * clamp(dot(attrib.normal, wi), 0.0f, 1.0f) / pdf / N;
		}
	}

	// ### NEE ###
    if (nee == NEE_ON) {

		// check for hitting light
		for (int i = 0; i < qlights.size(); i++) {
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float t = - (dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
			if (t < cf.epsilon && t > -cf.epsilon) { // hitting a light
				if (payload.depth == 0) {
					payload.radiance += mv.emission;
				}
				payload.done = true; 
				return; 
			}
		}

		// direct lighting
		float3 dlResult = mv.emission;
		for (int i = 0; i < qlights.size(); i++) {
			float3 tempResult = make_float3(0, 0, 0);

			float A = length(cross(qlights[i].ab, qlights[i].ac));
			float3 hp = attrib.intersection;
			float3 sn = normalize(attrib.normal);
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float3 wo = normalize(attrib.wo);
			float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
			int stepNum = (int)sqrt((float)lightSamples);
			for (int ls = 0; ls < lightSamples; ls++) {
				float3 lp;
				// randomize a light point
				if (lightStratify) {
					float3 abStep = qlights[i].ab / stepNum;
					float3 acStep = qlights[i].ac / stepNum;
					lp = qlights[i].a + (ls % stepNum) * abStep + (ls / stepNum) * acStep + rnd(payload.seed) * abStep + rnd(payload.seed) * acStep;
				}
				else {
					lp = qlights[i].a + rnd(payload.seed) * qlights[i].ab + rnd(payload.seed) * qlights[i].ac;
				}

				// check for shadow
				float3 lightDir = normalize(lp - hp);
				float lightDist = length(lp - hp);
				ShadowPayload shadowPayload;
				shadowPayload.isVisible = true;
				Ray shadowRay = make_Ray(hp, lightDir, 1, cf.epsilon, lightDist - cf.epsilon);
				rtTrace(root, shadowRay, shadowPayload);
				// If not in shadow
				if (shadowPayload.isVisible)
				{
					float3 wi = lightDir;
					// ### BRDF 2ND ###
					float3 f; 
					if (brdf == BRDF_PHONG) {
						f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) * pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
						float G = clamp(dot(sn, wi), 0.0f, 1.0f) * clamp(dot(ln, wi), 0.0f, 1.0f) / (lightDist * lightDist);
						f = f * G; 
					}
					else if (brdf == BRDF_GGX) {
						f = getGGXBRDF(attrib, wi);
					}
					tempResult += f;
				}
			}

			dlResult += qlights[i].color * A / lightSamples * tempResult;
		}

		// calculate radiance
		payload.radiance += payload.throughput * dlResult;
    }
    else if (nee == NEE_OFF) { 

		// check for hitting light
		for (int i = 0; i < qlights.size(); i++) {
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float t = -(dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
			if (t < cf.epsilon && t > -cf.epsilon) { // hitting a light
				payload.radiance += payload.throughput * mv.emission;
				payload.done = true;
				return;
			}
		}

        payload.radiance += payload.throughput * mv.emission;
	}
	else if (nee == NEE_MIS) {
		// TODO
	}

	// calculate Russian Roulette
	if (russianRoulette) {
		float q = 1 - fminf(fmaxf(throughput.x, fmaxf(throughput.y, throughput.z)), 1.0f);
		if (rnd(payload.seed) <= q) { //terminate
			payload.done = true;
			return;
		}
		else {
			float boost = 1.0f / (1.0f - q);
			throughput *= boost;
		}
	}
    
    // for recursion
    payload.origin = attrib.intersection;
    payload.dir = wi; 
    //add throughput for next iteration
    payload.throughput = payload.throughput * throughput;
    payload.depth++; 
}

float3 transformRay(float3 ray, float3 w, float epsilon) {
	float3 a = make_float3(0, 1, 0);
	if (length(w - a) < epsilon || length(w + a) < epsilon) {//avoid a too close to w
		a = make_float3(1, 0, 0);
	}
	float3 u = normalize(cross(a, w));
	float3 v = cross(w, u);
	return ray.x * u + ray.y * v + ray.z * w;
}

float3 getHemisphereSampleRay(float epsilon) {
	float3 wi; 
	float theta = acosf(rnd(payload.seed));
	float phi = 2 * M_PIf * rnd(payload.seed);
	float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
	float3 w = normalize(attrib.normal);
	wi = transformRay(s, w, epsilon);
	return wi; 
}

float3 getCosineSampleRay(float epsilon) {
	float3 wi; 
	float theta = acosf(sqrt(rnd(payload.seed)));
	float phi = 2 * M_PIf * rnd(payload.seed);
	float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
	float3 w = normalize(attrib.normal);
	wi = transformRay(s, w, epsilon); 
	return wi; 
}

float3 getBRDFSampleRay(Attributes attrib, float epsilon) {
	MaterialValue mv = attrib.mv; 
	float3 wi; 
	float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
	float ks = (mv.specular.x + mv.specular.y + mv.specular.z) / 3.0f;
	float kd = (mv.diffuse.x + mv.diffuse.y + mv.diffuse.z) / 3.0f;
	float t = ks / (ks + kd);

	float phi = 2 * M_PIf * rnd(payload.seed);
	float theta = 0;
	float3 s, w;
	if (rnd(payload.seed) <= t) { //specular
		theta = acosf(powf(rnd(payload.seed), 1 / (mv.shininess + 1)));
		s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
		w = rl;
	}
	else { // diffuse
		theta = acosf(sqrt(rnd(payload.seed)));
		s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
		w = normalize(attrib.normal);
	}
	wi = transformRay(s, w, epsilon); 
	return wi; 
}

float3 getPhongBRDF(Attributes attrib, float3 wi) {
	MaterialValue mv = attrib.mv; 
	float3 f; 
	float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
	f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) *
		pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
	return f; 
}

float3 getGGXBRDF(Attributes attrib, float3 wi) {
	MaterialValue mv = attrib.mv; 
	float3 n = attrib.normal;
	float3 h = normalize(wi + attrib.wo);
	float alpha_cube = roughness * roughness;
	float theta_h = acosf(dot(h, n));
	float D = alpha_cube / (M_PIf * powf(cosf(theta_h), 4) *
		powf((alpha_cube + tanf(theta_h) * tanf(theta_h)), 2));
	
	float theta_wi = acosf(dot(wi, n));
	float G1_wi = dot(wi, n) > 0 ?
		2.0f / (1 + sqrtf(1 + alpha_cube * tanf(theta_wi) * tanf(theta_wi))) : 0;
	float theta_wo = acosf(dot(attrib.wo, n));
	float G1_wo = dot(attrib.wo, n) > 0 ?
		2.0f / (1 + sqrtf(1 + alpha_cube * tanf(theta_wo) * tanf(theta_wo))) : 0;
	float G = G1_wi * G1_wo;

	float3 F = mv.specular + (1 - mv.specular) * powf((1 - dot(wi, h)), 5);
	float3 f_ggx = F * G * D / (4 * dot(wi, n) * dot(attrib.wo, n));
	float3 f = mv.diffuse / M_PIf + f_ggx;

	return f;
}

float getHemispherePDF() {
	return 1 / (2 * M_PIf);
}

float getCosinePDF() {
	return 1 / M_PIf; 
}

float getBRDFPDF(Attributes attrib, float3 wi) {
	MaterialValue mv = attrib.mv; 
	float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
	float ks = (mv.specular.x + mv.specular.y + mv.specular.z) / 3.0f;
	float kd = (mv.diffuse.x + mv.diffuse.y + mv.diffuse.z) / 3.0f;
	float t = ks / (ks + kd);
	float pdf = (1 - t) * clamp(dot(attrib.normal, wi), 0.0f, 1.0f) / M_PIf +
		t * (mv.shininess + 1) / (2 * M_PIf) * pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
	return pdf; 
}


float3 getGGXThroughput(Attributes attrib, float3& wi) {
	MaterialValue mv = attrib.mv;

	float ks = (mv.specular.x + mv.specular.y + mv.specular.z) / 3.0f;
	float kd = (mv.diffuse.x + mv.diffuse.y + mv.diffuse.z) / 3.0f;
	float t = fmaxf(0.25f, ks / (ks + kd));
	float3 n = attrib.normal;

	// sample
	if (rnd(payload.seed) <= t) { // specular
		float phi = 2 * M_PIf * rnd(payload.seed);
		float rand = rnd(payload.seed);
		float theta = atanf(roughness * sqrtf(rand) / sqrtf(1 - rand));
		float3 h = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
		h = transformRay(h, n, config[0].epsilon);
		wi = reflect(-attrib.wo, h);
	}
	else { // diffuse
		float phi = 2 * M_PIf * rnd(payload.seed);
		float theta = acosf(sqrtf(rnd(payload.seed)));
		float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
		wi = transformRay(s, n, config[0].epsilon);
	}

	// BRDF
	float3 h = normalize(wi + attrib.wo);
	float alpha_cube = roughness * roughness;
	float theta_h = acosf(dot(h, n));
	float D = alpha_cube / (M_PIf * powf(cosf(theta_h), 4) *
		powf((alpha_cube + tanf(theta_h) * tanf(theta_h)), 2));

	float theta_wi = acosf(dot(wi, n));
	float G1_wi = dot(wi, n) > 0 ?
		2.0f / (1 + sqrtf(1 + alpha_cube * tanf(theta_wi) * tanf(theta_wi))) : 0;
	float theta_wo = acosf(dot(attrib.wo, n));
	float G1_wo = dot(attrib.wo, n) > 0 ?
		2.0f / (1 + sqrtf(1 + alpha_cube * tanf(theta_wo) * tanf(theta_wo))) : 0;
	float G = G1_wi * G1_wo;

	float3 F = mv.specular + (1 - mv.specular) * powf((1 - dot(wi, h)), 5);
	float3 f_ggx = F * G * D / (4 * dot(wi, n) * dot(attrib.wo, n));
	float3 f = mv.diffuse / M_PIf + f_ggx;

	//PDF
	float pdf = (1 - t) * dot(n, wi) / M_PIf + t * D * dot(n, h) / (4 * dot(h, wi));

	float3 throughput = f * clamp(dot(attrib.normal, wi), 0.0f, 1.0f) / pdf;
	return throughput;
}