#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(int, lightSamples, , );
rtDeclareVariable(int, lightStratify, , );
rtDeclareVariable(int, nee, , );
rtDeclareVariable(int, russianRoulette, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );


RT_PROGRAM void pathTracer() {
    MaterialValue mv = attrib.mv;
    Config cf = config[0];
    //float3 result = mv.emission;

    // find wi
    float theta = acos(rnd(payload.seed));
    float phi = 2 * M_PIf * rnd(payload.seed);
    float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));
    float3 w = normalize(attrib.normal);
    float3 a = make_float3(0, 1, 0);
    if (length(w - a) < cf.epsilon || length(w + a) < cf.epsilon) {//avoid a too close to w
        a = make_float3(1, 0, 0);
    }
    float3 u = normalize(cross(a, w));
    float3 v = cross(w, u);
    float3 wi = s.x * u + s.y * v + s.z * w;

    float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
    float3 f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) * 
        pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
    float inv_pdf = 2 * M_PIf;
	int N = 1; 
    float3 throughput = f * clamp(dot(attrib.normal, wi), 0.0f, 1.0f) * inv_pdf / N;

    if (nee) {

		// check for hitting light
		for (int i = 0; i < qlights.size(); i++) {
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float t = - (dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
			if (t < cf.epsilon && t > -cf.epsilon) { // hitting a light
				if (payload.depth == 0) {
					payload.radiance += mv.emission;
				}
				payload.done = true; 
				return; 
			}
		}

		// direct lighting
		float3 dlResult = mv.emission;
		for (int i = 0; i < qlights.size(); i++) {
			float3 tempResult = make_float3(0, 0, 0);

			float A = length(cross(qlights[i].ab, qlights[i].ac));
			float3 hp = attrib.intersection;
			float3 sn = normalize(attrib.normal);
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float3 wo = normalize(attrib.wo);
			float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
			int stepNum = (int)sqrt((float)lightSamples);
			for (int ls = 0; ls < lightSamples; ls++) {
				float3 lp;
				// randomize a light point
				if (lightStratify) {
					float3 abStep = qlights[i].ab / stepNum;
					float3 acStep = qlights[i].ac / stepNum;
					lp = qlights[i].a + (ls % stepNum) * abStep + (ls / stepNum) * acStep + rnd(payload.seed) * abStep + rnd(payload.seed) * acStep;
				}
				else {
					lp = qlights[i].a + rnd(payload.seed) * qlights[i].ab + rnd(payload.seed) * qlights[i].ac;
				}

				// check for shadow
				float3 lightDir = normalize(lp - hp);
				float lightDist = length(lp - hp);
				ShadowPayload shadowPayload;
				shadowPayload.isVisible = true;
				Ray shadowRay = make_Ray(hp, lightDir, 1, cf.epsilon, lightDist - cf.epsilon);
				rtTrace(root, shadowRay, shadowPayload);
				// If not in shadow
				if (shadowPayload.isVisible)
				{
					float3 wi = lightDir;
					//rtPrintf("ln.x : %f, y: %f, z: %f\n", ln.x, ln.y, ln.z); 
					float3 f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) * pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
					float G = clamp(dot(sn, wi), 0.0f, 1.0f) * clamp(dot(ln, wi), 0.0f, 1.0f) / (lightDist * lightDist);
					tempResult += f * G;
				}
			}

			dlResult += qlights[i].color * A / lightSamples * tempResult;
		}

		// calculate radiance
		payload.radiance += payload.throughput * dlResult;
    }
    else { // not nee

		// check for hitting light
		for (int i = 0; i < qlights.size(); i++) {
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float t = -(dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
			if (t < cf.epsilon && t > -cf.epsilon) { // hitting a light
				payload.radiance += payload.throughput * mv.emission;
				payload.done = true;
				return;
			}
		}

        payload.radiance += payload.throughput * mv.emission;
    }

	// calculate Russian Roulette
	if (russianRoulette) {
		float q = 1 - fminf(fmaxf(throughput.x, fmaxf(throughput.y, throughput.z)), 1.0f);
		if (rnd(payload.seed) <= q) { //terminate
			payload.done = true;
			return;
		}
		else {
			float boost = 1.0f / (1.0f - q);
			throughput *= boost;
		}
	}

    
    // for recursion
    payload.origin = attrib.intersection;
    payload.dir = wi; 
    //add throughput for next iteration
    payload.throughput = payload.throughput * throughput;
    payload.depth++; 
}


