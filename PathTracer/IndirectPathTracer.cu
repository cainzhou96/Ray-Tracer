#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

#define IS_HEMISPEHRE 0
#define IS_COSINE 1
#define IS_BRDF 2

#define NEE_OFF 0
#define NEE_ON 1
#define NEE_MIS 2

#define BRDF_PHONG 0
#define BRDF_GGX 1

using namespace optix;

// Declare light buffers
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(int, lightSamples, , );
rtDeclareVariable(int, lightStratify, , );
rtDeclareVariable(int, nee, , );
rtDeclareVariable(int, importanceSampling, , );
rtDeclareVariable(int, russianRoulette, , );

rtDeclareVariable(int, brdf, , );
rtDeclareVariable(int, roughness, , );
rtDeclareVariable(int, gamma, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );


RT_PROGRAM void pathTracer() {
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

	// ### SAMPLE ###
	float3 wi; 
	if (importanceSampling == IS_HEMISPEHRE) {
		float theta = acos(rnd(payload.seed));
		float phi = 2 * M_PIf * rnd(payload.seed);
		float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));
		float3 w = normalize(attrib.normal);
		float3 a = make_float3(0, 1, 0);
		if (length(w - a) < cf.epsilon || length(w + a) < cf.epsilon) {//avoid a too close to w
			a = make_float3(1, 0, 0);
		}
		float3 u = normalize(cross(a, w));
		float3 v = cross(w, u);
		wi = s.x * u + s.y * v + s.z * w;
	}
	else if (importanceSampling == IS_COSINE) {
		float theta = acos(sqrt(rnd(payload.seed)));
		float phi = 2 * M_PIf * rnd(payload.seed);
		float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));
		float3 w = normalize(attrib.normal);
		float3 a = make_float3(0, 1, 0);
		if (length(w - a) < cf.epsilon || length(w + a) < cf.epsilon) {//avoid a too close to w
			a = make_float3(1, 0, 0);
		}
		float3 u = normalize(cross(a, w));
		float3 v = cross(w, u);
		wi = s.x * u + s.y * v + s.z * w;
	}
	else if (importanceSampling == IS_BRDF) {
		// TODO
	}

	// ### BRDF ###
	float3 f; 
	if (brdf == BRDF_PHONG) {
		float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
		f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) * 
			pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
	}
	else if (brdf == BRDF_GGX) {
		// TODO
	}

	// ### PDF ###
	float inv_pdf; 
	int N; 
	float3 throughput; 
	if (importanceSampling == IS_HEMISPEHRE) {
		inv_pdf = 2 * M_PIf;
		N = 1; 
		throughput = f * clamp(dot(attrib.normal, wi), 0.0f, 1.0f) * inv_pdf / N;
	}
	else if (importanceSampling == IS_COSINE) {
		inv_pdf = M_PIf;
		N = 1; 
		throughput = f * inv_pdf / N;
	}
	else if (importanceSampling == IS_BRDF) {
		// TODO
	}

	// ### NEE ###
    if (nee == NEE_ON) {

		// check for hitting light
		for (int i = 0; i < qlights.size(); i++) {
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float t = - (dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
			if (t < cf.epsilon && t > -cf.epsilon) { // hitting a light
				if (payload.depth == 0) {
					payload.radiance += mv.emission;
				}
				payload.done = true; 
				return; 
			}
		}

		// direct lighting
		float3 dlResult = mv.emission;
		for (int i = 0; i < qlights.size(); i++) {
			float3 tempResult = make_float3(0, 0, 0);

			float A = length(cross(qlights[i].ab, qlights[i].ac));
			float3 hp = attrib.intersection;
			float3 sn = normalize(attrib.normal);
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float3 wo = normalize(attrib.wo);
			float3 rl = normalize(reflect(-attrib.wo, attrib.normal));
			int stepNum = (int)sqrt((float)lightSamples);
			for (int ls = 0; ls < lightSamples; ls++) {
				float3 lp;
				// randomize a light point
				if (lightStratify) {
					float3 abStep = qlights[i].ab / stepNum;
					float3 acStep = qlights[i].ac / stepNum;
					lp = qlights[i].a + (ls % stepNum) * abStep + (ls / stepNum) * acStep + rnd(payload.seed) * abStep + rnd(payload.seed) * acStep;
				}
				else {
					lp = qlights[i].a + rnd(payload.seed) * qlights[i].ab + rnd(payload.seed) * qlights[i].ac;
				}

				// check for shadow
				float3 lightDir = normalize(lp - hp);
				float lightDist = length(lp - hp);
				ShadowPayload shadowPayload;
				shadowPayload.isVisible = true;
				Ray shadowRay = make_Ray(hp, lightDir, 1, cf.epsilon, lightDist - cf.epsilon);
				rtTrace(root, shadowRay, shadowPayload);
				// If not in shadow
				if (shadowPayload.isVisible)
				{
					float3 wi = lightDir;
					// ### BRDF 2ND ###
					float3 f; 
					if (brdf == BRDF_PHONG) {
						f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) * pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
						float G = clamp(dot(sn, wi), 0.0f, 1.0f) * clamp(dot(ln, wi), 0.0f, 1.0f) / (lightDist * lightDist);
						f = f * G; 
					}
					else if (brdf == BRDF_GGX) {
						// TODO
					}
					tempResult += f;
				}
			}

			dlResult += qlights[i].color * A / lightSamples * tempResult;
		}

		// calculate radiance
		payload.radiance += payload.throughput * dlResult;
    }
    else if (nee == NEE_OFF) { 

		// check for hitting light
		for (int i = 0; i < qlights.size(); i++) {
			float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac));
			float t = -(dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
			if (t < cf.epsilon && t > -cf.epsilon) { // hitting a light
				payload.radiance += payload.throughput * mv.emission;
				payload.done = true;
				return;
			}
		}

        payload.radiance += payload.throughput * mv.emission;
	}
	else if (nee == NEE_MIS) {
		// TODO
	}

	// calculate Russian Roulette
	if (russianRoulette) {
		float q = 1 - fminf(fmaxf(throughput.x, fmaxf(throughput.y, throughput.z)), 1.0f);
		if (rnd(payload.seed) <= q) { //terminate
			payload.done = true;
			return;
		}
		else {
			float boost = 1.0f / (1.0f - q);
			throughput *= boost;
		}
	}
    
    // for recursion
    payload.origin = attrib.intersection;
    payload.dir = wi; 
    //add throughput for next iteration
    payload.throughput = payload.throughput * throughput;
    payload.depth++; 
}


