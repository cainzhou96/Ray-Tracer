#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

#define IS_HEMISPEHRE 0
#define IS_COSINE 1
#define IS_BRDF 2

#define NEE_OFF 0
#define NEE_ON 1
#define NEE_MIS 2

#define BRDF_PHONG 0
#define BRDF_GGX 1

using namespace optix;

// Declare light buffers
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(int, lightSamples, , );
rtDeclareVariable(int, lightStratify, , );
rtDeclareVariable(int, nee, , );
rtDeclareVariable(int, importanceSampling, , );
rtDeclareVariable(int, russianRoulette, , );

rtDeclareVariable(float, gamma, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

float3 transformRay(float3 ray, float3 w); 
float3 getCosineSampleRay(); 
float3 getHemisphereSampleRay(); 
float3 getBRDFSampleRay(); 

float3 getPhongBRDF(float3 wi); 
float3 getGGXBRDF(float3 wi); 

float getCosinePDF(); 
float getHemispherePDF(); 
float getBRDFPDF(float3 wi); 
float getNeePDF(float3 wi); 

float3 getNEEDirectLighting(); 
float3 getBRDFDirectLighting(float3 wi); 

int isHittingLight(); 
float power(float base, int exp);
void printF3(float3 v); 

RT_PROGRAM void pathTracer() {
    MaterialValue mv = attrib.mv;
    Config cf = config[0];
	float3 wi;
	float3 throughput;

	// ### SAMPLE ###
	if (importanceSampling == IS_HEMISPEHRE) {
		wi = getHemisphereSampleRay();
	}
	else if (importanceSampling == IS_COSINE) {
		wi = getCosineSampleRay();
	}
	else if (importanceSampling == IS_BRDF) {
		wi = getBRDFSampleRay();
	}

	// ### BRDF ###
	float3 f; 
	if (mv.brdf == BRDF_PHONG) {
		f = getPhongBRDF(wi);
	}
	else if (mv.brdf == BRDF_GGX) {
		f = getGGXBRDF(wi);
	}

	// ### PDF ###
	float pdf;
	int N = 1;
	float3 n = normalize(attrib.normal); 
	if (importanceSampling == IS_HEMISPEHRE) {
		pdf = getHemispherePDF();
		throughput = f * clamp(dot(n, wi), 0.0f, 1.0f) / pdf / N;
	}
	else if (importanceSampling == IS_COSINE) {
		pdf = getCosinePDF();
		throughput = f / pdf / N;
	}
	else if (importanceSampling == IS_BRDF) {
		pdf = getBRDFPDF(wi);
		if (pdf <= 0) {
			throughput = make_float3(0, 0, 0); 
		}
		else {
			throughput = f * clamp(dot(n, wi), 0.0f, 1.0f) / pdf / N;
		}
	}

	// ### NEE ###
    if (nee == NEE_ON) {

		// check for hitting light
		int lightHit = isHittingLight(); 
		if (lightHit == 1) { // hit front 
			if (payload.depth == 0) {
				payload.radiance += payload.throughput * mv.emission;
			}
			payload.done = true; 
			return; 
		}
		else if (lightHit == 2) { // hit back
			payload.done = true;
			return;
		}

		// direct lighting result
		float3 dlResult = getNEEDirectLighting();

		// calculate radiance
		payload.radiance += payload.throughput * dlResult;
    }
    else if (nee == NEE_OFF) { 

		// check for hitting light
		int lightHit = isHittingLight();
		if (lightHit == 1) { // hit front 
			payload.radiance += payload.throughput * mv.emission;
			payload.done = true;
			return;
		}
		else if (lightHit == 2) { // hit back
			payload.done = true;
			return;
		}

        payload.radiance += payload.throughput * mv.emission;
	}
	else if (nee == NEE_MIS) {
		// check for hitting light
		int lightHit = isHittingLight();
		if (lightHit == 1) { // hit front 
			if (payload.depth == 0) {
				payload.radiance += payload.throughput * mv.emission;
			}
			payload.done = true;
			return;
		}
		else if (lightHit == 2) { // hit back
			payload.done = true;
			return;
		}

		int beta = 2; 
		float3 DLResult = mv.emission; 
		float3 curWi; 
		float3 curDLResult; 
		float3 curF; 
		float3 curThroughput; 
		float curBRDFPDF; 
		float curNEEPDF; 
		float curWeight; 

		// brdf
		curWi = getBRDFSampleRay();
		curDLResult = getBRDFDirectLighting(curWi);
		curBRDFPDF = getBRDFPDF(curWi); 
		curNEEPDF = getNeePDF(curWi); 
		n = normalize(attrib.normal); 
		if (curBRDFPDF <= 0) {
			curThroughput = make_float3(0); 
		}
		else {
			curWeight = power(curBRDFPDF, beta) / (power(curBRDFPDF, beta) + power(curNEEPDF, beta)); 
			if (mv.brdf == BRDF_PHONG) {
				curF = getPhongBRDF(curWi); 
			}
			else if (mv.brdf == BRDF_GGX) {
				curF = getGGXBRDF(curWi); 
			}
			curThroughput = curWeight * curF * clamp(dot(n, curWi), 0.0f, 1.0f) / curBRDFPDF; 
		}
		DLResult += curThroughput * curDLResult; 

		// nee
		curDLResult = make_float3(0);
		for (int i = 0; i < qlights.size(); i++) {
			float A = length(cross(qlights[i].ab, qlights[i].ac));
			float3 hp = attrib.intersection;
			float3 sn = normalize(attrib.normal);
			float3 ln = -normalize(cross(qlights[i].ab, qlights[i].ac));
			float3 wo = normalize(attrib.wo);
			float3 rl = normalize(reflect(-wo, sn));
			int stepNum = (int)sqrt((float)lightSamples);
			for (int ls = 0; ls < lightSamples; ls++) {
				float3 lp;
				// randomize a light point
				if (lightStratify) {
					float3 abStep = qlights[i].ab / stepNum;
					float3 acStep = qlights[i].ac / stepNum;
					lp = qlights[i].a + (ls % stepNum) * abStep + (ls / stepNum) * acStep + rnd(payload.seed) * abStep + rnd(payload.seed) * acStep;
				}
				else {
					lp = qlights[i].a + rnd(payload.seed) * qlights[i].ab + rnd(payload.seed) * qlights[i].ac;
				}

				// check for shadow
				float3 lightDir = normalize(lp - hp);
				float lightDist = length(lp - hp);
				ShadowPayload shadowPayload;
				shadowPayload.isVisible = true;
				Ray shadowRay = make_Ray(hp, lightDir, 1, config[0].epsilon, lightDist - config[0].epsilon);
				rtTrace(root, shadowRay, shadowPayload);
				// If not in shadow
				if (shadowPayload.isVisible)
				{
					curDLResult = qlights[i].color; 
					curWi = lightDir; 
					curBRDFPDF = getBRDFPDF(curWi); 
					curNEEPDF = getNeePDF(curWi); 
					if (curNEEPDF == 0) {
						curThroughput = make_float3(0); // hack it for now
					}
					else {
						if (curBRDFPDF <= 0) {
							curBRDFPDF = 0; 
						}
						curWeight = power(curNEEPDF, beta) / (power(curBRDFPDF, beta) + power(curNEEPDF, beta)); 
						if (mv.brdf == BRDF_PHONG) {
							curF = getPhongBRDF(curWi); 
						}
						else if (mv.brdf == BRDF_GGX) {
							curF = getGGXBRDF(curWi); 
						}
						float G = clamp(dot(sn, curWi), 0.0f, 1.0f) * clamp(dot((-ln), curWi), 0.0f, 1.0f) / (lightDist * lightDist);
						curThroughput = curWeight * curF * G * A / lightSamples; 
						DLResult += curThroughput * curDLResult; 
					}
				}
			}
		}
		payload.radiance += payload.throughput * DLResult;
	}

	// calculate Russian Roulette
	if (russianRoulette) {
		float q = 1 - fminf(fmaxf(throughput.x, fmaxf(throughput.y, throughput.z)), 1.0f);
		if (rnd(payload.seed) <= q) { //terminate
			payload.done = true;
			return;
		}
		else {
			float boost = 1.0f / (1.0f - q);
			throughput *= boost;
		}
	}
    
    // for recursion
    payload.origin = attrib.intersection;
    payload.dir = wi; 
    //add throughput for next iteration
    payload.throughput = payload.throughput * throughput;
    payload.depth++; 
}

float3 transformRay(float3 ray, float3 w) {
	float3 a = make_float3(0, 1, 0);
	if (length(w - a) < config[0].epsilon || length(w + a) < config[0].epsilon) {//avoid a too close to w
		a = make_float3(1, 0, 0);
	}
	float3 u = normalize(cross(a, w));
	float3 v = cross(w, u);
	return ray.x * u + ray.y * v + ray.z * w;
}

float3 getHemisphereSampleRay() {
	float3 wi; 
	float theta = acosf(rnd(payload.seed));
	float phi = 2 * M_PIf * rnd(payload.seed);
	float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
	float3 w = normalize(attrib.normal);
	wi = transformRay(s, w);
	wi = normalize(wi); 
	return wi; 
}

float3 getCosineSampleRay() {
	float3 wi; 
	float theta = acosf(sqrt(rnd(payload.seed)));
	float phi = 2 * M_PIf * rnd(payload.seed);
	float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
	float3 w = normalize(attrib.normal);
	wi = transformRay(s, w); 
	wi = normalize(wi);
	return wi; 
}

float3 getBRDFSampleRay() {
	MaterialValue mv = attrib.mv; 
	float3 wi; 
	float3 wo = normalize(attrib.wo); 
	float3 n = normalize(attrib.normal); 
	float ks = (mv.specular.x + mv.specular.y + mv.specular.z) / 3.0f;
	float kd = (mv.diffuse.x + mv.diffuse.y + mv.diffuse.z) / 3.0f;
	if (mv.brdf == BRDF_PHONG) {
		float3 rl = normalize(-reflect(wo, n));
		float t = ks / (ks + kd);

		float phi = 2 * M_PIf * rnd(payload.seed);
		float theta = 0;
		float3 s, w;
		if (rnd(payload.seed) <= t) { //specular
			theta = acosf(power(rnd(payload.seed), 1 / (mv.shininess + 1)));
			s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
			w = rl;
		}
		else { // diffuse
			theta = acosf(sqrtf(rnd(payload.seed)));
			s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
			w = normalize(n);
		}
		wi = transformRay(s, w); 
	}
	else if (mv.brdf == BRDF_GGX) {
		float t = fmaxf(0.25f, ks / (ks + kd));
		float phi = 2 * M_PIf * rnd(payload.seed);

		if (rnd(payload.seed) <= t) { // specular
			float rand = rnd(payload.seed);
			float theta = atanf(mv.roughness * sqrtf(rand) / sqrtf(1 - rand));
			float3 h = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
			float3 w = n;
			h = transformRay(h, w);
			wi = reflect(-wo, h);
		}
		else { // diffuse
			float theta = acosf(sqrtf(rnd(payload.seed)));
			float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
			float3 w = n;
			wi = transformRay(s, w);
		}
	}
	wi = normalize(wi);
	return wi; 
}

float3 getPhongBRDF(float3 wi) {
	MaterialValue mv = attrib.mv; 
	float3 wo = normalize(attrib.wo); 
	float3 n = normalize(attrib.normal); 
	float3 rl = normalize(reflect(-wo, n));
	float3 f; 
	f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) *
		power(dot(rl, wi), mv.shininess);
	return f; 
}

float3 getGGXBRDF(float3 wi) {
	MaterialValue mv = attrib.mv; 
	float3 wo = normalize(attrib.wo);
	float3 n = normalize(attrib.normal);
	float3 h = normalize(wi + wo);
	float3 f_ggx;
	float3 f; 
	if (dot(wi, n) <= 0 || dot(wo, n) <= 0) {
		f_ggx = make_float3(0, 0, 0);
	}
	else {
		float alpha_square = mv.roughness * mv.roughness;
		float theta_h; 
		if (dot(h, n) < 1) {
			theta_h = acosf(dot(h, n));
		}
		else {
			theta_h = 0; 
		}
		float D = alpha_square / (M_PIf * power(cosf(theta_h), 4) *
			power((alpha_square + tanf(theta_h) * tanf(theta_h)), 2));

		float theta_wi = acosf(dot(wi, n));
		float G1_wi = 2.0f / (1 + sqrtf(1 + alpha_square * tanf(theta_wi) * tanf(theta_wi)));
		float theta_wo = acosf(dot(wo, n));
		float G1_wo = 2.0f / (1 + sqrtf(1 + alpha_square * tanf(theta_wo) * tanf(theta_wo)));
		float G = G1_wi * G1_wo;

		float3 F = mv.specular + (make_float3(1.0f, 1.0f, 1.0f) - mv.specular) * power((1 - dot(wi, h)), 5);
		f_ggx = F * G * D / (4 * dot(wi, n) * dot(wo, n));
	}
	f = mv.diffuse / M_PIf + f_ggx;
	return f;
}

float getHemispherePDF() {
	return 1 / (2 * M_PIf);
}

float getCosinePDF() {
	return 1 / M_PIf; 
}

float getBRDFPDF(float3 wi) {
	MaterialValue mv = attrib.mv; 
	float3 wo = normalize(attrib.wo);
	float3 n = normalize(attrib.normal); 
	float3 rl = normalize(reflect(-wo, n));
	float ks = (mv.specular.x + mv.specular.y + mv.specular.z) / 3.0f;
	float kd = (mv.diffuse.x + mv.diffuse.y + mv.diffuse.z) / 3.0f;
	float pdf; 
	if (mv.brdf == BRDF_PHONG) {
		float t = ks / (ks + kd);
		if (isnan(t))
			t = 0;
		pdf = (1 - t) * clamp(dot(n, wi), 0.0f, 1.0f) / M_PIf +
			t * (mv.shininess + 1) / (2 * M_PIf) * power(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
	}
	else if (mv.brdf == BRDF_GGX) {
		float t = fmaxf(0.25f, ks / (ks + kd));
		float3 h = normalize(wi + wo);
		float alpha_square = mv.roughness * mv.roughness;
		float theta_h; 
		if (dot(h, n) < 1) {
			theta_h = acosf(dot(h, n));
		}
		else {
			theta_h = 0; 
		}
		float D = alpha_square / (M_PIf * power(cosf(theta_h), 4) *
			power((alpha_square + tanf(theta_h) * tanf(theta_h)), 2));
		pdf = (1 - t) * clamp(dot(n, wi), 0.0f, 1.0f) / M_PIf + t * D * dot(n, h) / (4 * dot(h, wi));
	}
	return pdf; 
}


float getNeePDF(float3 wi) {
	if (qlights.size() == 0)
		return 0;

	float pdf_nee = 0;
	// check for hitting light
	for (int i = 0; i < qlights.size(); i++) {
		QuadLight q = qlights[i];
		float3 ln = -normalize(cross(qlights[i].ab, qlights[i].ac));
		float t = dot(qlights[i].a - attrib.intersection, ln) / dot(wi, ln);
		if (t > 0) {
			float3 hp = attrib.intersection + wi * t;
			float u = dot(hp - q.a, q.ab);
			float v = dot(hp - q.a, q.ac);
			// hit quad light (MAYBE INCORRECT)
			float3 ab = qlights[i].ab + config[0].epsilon * normalize(qlights[i].ab);
			float3 ac = qlights[i].ac + config[0].epsilon * normalize(qlights[i].ac);
			if (u >= 0 && u <= dot(ab, ab) && v >= 0 && v <= dot(ac, ac)) {
				float A = length(cross(qlights[i].ab, qlights[i].ac));
				float R = fabsf(t);
				pdf_nee += R * R / (A * fabsf(dot(ln, wi)));
			}
		}
	}
	pdf_nee = pdf_nee / qlights.size();
	return pdf_nee;

	/*
	if (qlights.size() == 0)
		return 0;

	float pdf_nee = 0;
	LightPayload lightPayload;
	lightPayload.hit = 0;
	lightPayload.emission = make_float3(0);
	Ray ray = make_Ray(attrib.intersection, wi, 2, config[0].epsilon, RT_DEFAULT_MAX);
	rtTrace(root, ray, lightPayload);
	if (!lightPayload.hit) {
		return 0; 
	}
	for (int i = 0; i < qlights.size(); i++) {
		float cur_pdf = 0; 
		float3 ln = -normalize(cross(qlights[i].ab, qlights[i].ac));
		float t = -(dot(qlights[i].a, ln) - dot(lightPayload.intersection, ln));
		if (t < config[0].epsilon && t > -config[0].epsilon) { // hitting a light
			float A = length(cross(qlights[i].ab, qlights[i].ac));
			float R = length(lightPayload.intersection - attrib.intersection);
			cur_pdf = R * R / A / abs(dot(ln, wi)); 
		}
		pdf_nee += cur_pdf; 
	}

	pdf_nee = pdf_nee / qlights.size();
	return pdf_nee;
	*/
}

float3 getNEEDirectLighting() {
	MaterialValue mv = attrib.mv; 
	float3 dlResult = mv.emission;
	for (int i = 0; i < qlights.size(); i++) {
		float3 tempResult = make_float3(0, 0, 0);

		float A = length(cross(qlights[i].ab, qlights[i].ac));
		float3 hp = attrib.intersection;
		float3 sn = normalize(attrib.normal);
		float3 ln = -normalize(cross(qlights[i].ab, qlights[i].ac));
		float3 wo = normalize(attrib.wo);
		float3 rl = normalize(reflect(-wo, sn));
		int stepNum = (int)sqrt((float)lightSamples);
		for (int ls = 0; ls < lightSamples; ls++) {
			float3 lp;
			// randomize a light point
			if (lightStratify) {
				float3 abStep = qlights[i].ab / stepNum;
				float3 acStep = qlights[i].ac / stepNum;
				lp = qlights[i].a + (ls % stepNum) * abStep + (ls / stepNum) * acStep + rnd(payload.seed) * abStep + rnd(payload.seed) * acStep;
			}
			else {
				lp = qlights[i].a + rnd(payload.seed) * qlights[i].ab + rnd(payload.seed) * qlights[i].ac;
			}

			// check for shadow
			float3 lightDir = normalize(lp - hp);
			float lightDist = length(lp - hp);
			ShadowPayload shadowPayload;
			shadowPayload.isVisible = true;
			Ray shadowRay = make_Ray(hp, lightDir, 1, config[0].epsilon, lightDist - config[0].epsilon);
			rtTrace(root, shadowRay, shadowPayload);
			// If not in shadow
			if (shadowPayload.isVisible)
			{
				// ### BRDF 2ND ###
				float3 f;
				float G = clamp(dot(sn, lightDir), 0.0f, 1.0f) * clamp(dot((-ln), lightDir), 0.0f, 1.0f) / (lightDist * lightDist);
				if (mv.brdf == BRDF_PHONG) {
					f = getPhongBRDF(lightDir);
					f = f * G;
				}
				else if (mv.brdf == BRDF_GGX) {
					f = getGGXBRDF(lightDir);
					f = f * G;
				}
				tempResult += f;
			}
		}

		dlResult += qlights[i].color * A / lightSamples * tempResult;
	}
	return dlResult; 
}

float3 getBRDFDirectLighting(float3 wi) {
	MaterialValue mv = attrib.mv;
	LightPayload lightPayload;
	lightPayload.hit = 0; 
	lightPayload.emission = make_float3(0);
	Ray ray = make_Ray(attrib.intersection, wi, 2, config[0].epsilon, RT_DEFAULT_MAX);
	rtTrace(root, ray, lightPayload);
	return lightPayload.emission; 
}

int isHittingLight() { // return 0: not hitting light, 1: hitting front, 2: hitting back
	for (int i = 0; i < qlights.size(); i++) {
		float3 ln = -normalize(cross(qlights[i].ab, qlights[i].ac));
		float t = -(dot(qlights[i].a, ln) - dot(attrib.intersection, ln));
		if (t < config[0].epsilon && t > -config[0].epsilon) { // hitting a light
			if (dot(ln, normalize(attrib.wo)) > 0) {
				return 1; 
			}
			else {
				return 2;
			}
		}
	}
	return 0; 
}

float power(float base, int exp) {
	if (exp == 0)
		return 1.0f;
	float res = base;
	for (int i = 1; i < exp; i++) {
		res *= base;
	}
	return res;
}

void printF3(float3 v) {
	rtPrintf("%f, %f, %f\n", v.x, v.y, v.z); 
}

