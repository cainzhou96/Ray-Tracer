#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon,
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon,
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}