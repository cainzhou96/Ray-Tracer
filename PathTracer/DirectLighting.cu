#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(int, lightSamples, , );
rtDeclareVariable(int, lightStratify, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void analytic()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;
    float3 f = mv.diffuse / M_PIf;

    for (int i = 0; i < qlights.size(); i++) {
        float3 v[4];
        v[0] = qlights[i].a;
        v[1] = qlights[i].a + qlights[i].ab;
        v[2] = qlights[i].a + qlights[i].ab + qlights[i].ac;
        v[3] = qlights[i].a + qlights[i].ac;
        float3 r = attrib.intersection;
        float3 phi = make_float3(0, 0, 0);
        for (int k = 0; k < 4; k++) {
            float3 v_next = v[0];
            if (k < 3)
                v_next = v[k + 1];
            float theta = acos(dot(normalize(v[k] - r), normalize(v_next - r)));
            float3 gamma = normalize(cross((v[k] - r), (v_next - r)));
            phi += theta * gamma;
        }
        phi *= 0.5f;
        result += f * qlights[i].color * dot(phi, attrib.normal);
    }


    // Compute the final radiance
    payload.radiance = result * payload.throughput;
    payload.done = true;
    
}

RT_PROGRAM void monteCarlo()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;
    for (int i = 0; i < qlights.size(); i++) {
        float3 tempResult = make_float3(0, 0, 0); 

        float A = length(cross(qlights[i].ab, qlights[i].ac)); 
        float3 hp = attrib.intersection;
        float3 sn = normalize(attrib.normal); 
        float3 ln = normalize(cross(qlights[i].ab, qlights[i].ac)); // ?
        float3 wo = normalize(attrib.wo); 
        float3 rl = normalize(reflect(-attrib.wo, attrib.normal)); // ?
        int stepNum = (int)sqrt((float)lightSamples); 
        for (int N = 0; N < lightSamples; N++) {
            float3 lp; 
            // randomize a light point
            if (lightStratify) {
                float3 abStep = qlights[i].ab / stepNum; 
                float3 acStep = qlights[i].ac / stepNum; 
                //rtPrintf("x : %f, y: %f, z: %f\n", abStep.x, abStep.y, abStep.z); 
                lp = qlights[i].a + (N % stepNum) * abStep + (N / stepNum) * acStep + rnd(payload.seed) * abStep + rnd(payload.seed) * acStep; 
                //rtPrintf("lp.x : %f, y: %f, z: %f\n", lp.x, lp.y, lp.z); 
            } else {
                lp = qlights[i].a + rnd(payload.seed) * qlights[i].ab + rnd(payload.seed) * qlights[i].ac; 
            }

            // check for shadow
            float3 lightDir = normalize(lp - hp);
            float lightDist = length(lp - hp);
            ShadowPayload shadowPayload;
            shadowPayload.isVisible = true;
            Ray shadowRay = make_Ray(hp, lightDir, 1, cf.epsilon, lightDist - cf.epsilon); // post @217
            rtTrace(root, shadowRay, shadowPayload);
            // If not in shadow
            if (shadowPayload.isVisible)
            {
                float3 wi = lightDir; 
                //rtPrintf("ln.x : %f, y: %f, z: %f\n", ln.x, ln.y, ln.z); 
                float3 f = mv.diffuse / M_PIf + mv.specular * (mv.shininess + 2) / (2 * M_PIf) * pow(clamp(dot(rl, wi), 0.0f, 1.0f), mv.shininess);
                float G = clamp(dot(sn, wi), 0.0f, 1.0f) * clamp(dot(ln, wi), 0.0f, 1.0f) / (lightDist * lightDist); 
                tempResult += f * G; 
            }
        }

        result += qlights[i].color * A / lightSamples * tempResult; 
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;
    payload.done = true;

}