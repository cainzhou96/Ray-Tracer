#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Config.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

rtBuffer<Config> config; // Config

RT_PROGRAM void generateRays()
{
    size_t2 resultSize = resultBuffer.size();
    unsigned int index = launchIndex.x * resultSize.y + launchIndex.y;
    unsigned int seed = tea<16>(index * frameID.x, 0);
    Config cf = config[0];
    float3 result = make_float3(0.f);

    // Compute the ray direction
    float2 xy = make_float2(launchIndex);
    xy.x += frameID.x == 1 ? 0.5f : rnd(seed);
    xy.y += frameID.x == 1 ? 0.5f : rnd(seed);
    float2 ab = cf.tanHFov * (xy - cf.hSize) / cf.hSize;
    float3 dir = normalize(ab.x * cf.u + ab.y * cf.v - cf.w); // ray direction
    float3 origin = cf.eye; // ray origin

    // Prepare a payload
    Payload payload;
    payload.radiance = make_float3(0.f);
    payload.throughput = make_float3(1.f);
    payload.depth = 0;
    payload.done = false;
    int i = 0;

    // Iteratively trace rays (recursion is very expensive on GPU)
    do
    {
        payload.seed = tea<16>(index * frameID.x, i++);

        // Trace a ray
        Ray ray = make_Ray(origin, dir, 0, cf.epsilon, RT_DEFAULT_MAX);
        rtTrace(root, ray, payload);

        // Accumulate radiance
        result += payload.radiance;
        payload.radiance = make_float3(0.f);

        // Prepare to shoot next ray
        origin = payload.origin;
        dir = payload.dir;
    } while (!payload.done && payload.depth != cf.maxDepth);
    
    if (frameID.x == 1) 
        resultBuffer[launchIndex] = result;
    else
    {
        float u = 1.0f / (float)frameID.x;
        float3 oldResult = resultBuffer[launchIndex];
        resultBuffer[launchIndex] = lerp(oldResult, result, u);
    }
}