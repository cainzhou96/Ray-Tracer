#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"

using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float1, epsilon, , ); // scene epsilon

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    Matrix4x4 itrans = sphere.trans.inverse();
    float4 rayOriH = itrans * make_float4(ray.origin, 1);
    float3 rayOri = make_float3(rayOriH) / rayOriH.w;
    float3 rayDir = normalize(make_float3(itrans * make_float4(ray.direction, 0)));

    float t = 0;
    float3 CP0 = rayOri;
    float P1dotCP0 = dot(rayDir, CP0);
    float CP0dotCP0 = dot(CP0, CP0);
    float r2 = 1.f;
    float disc = P1dotCP0 * P1dotCP0 - CP0dotCP0 + r2;
    if (disc < 0) return;
    if (disc == 0)
        t = -P1dotCP0;
    else if (CP0dotCP0 > r2)
        t = -P1dotCP0 - sqrt(disc);
    else
        t = -P1dotCP0 + sqrt(disc);

    if (t < epsilon.x) return;

    // Intersection is found
    float3 P = rayOri + t * rayDir; // intersection in the object space
    float4 intersectionH = sphere.trans * make_float4(P, 1);
    P = make_float3(intersectionH) / intersectionH.w; // intersection in the world space
    t = length(P - ray.origin); // distance

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // Pass attributes
        attrib.intersection = P;
        attrib.wo = -ray.direction;
        float4 tintersection = sphere.trans.inverse() * make_float4(attrib.intersection, 1);
        attrib.normal = normalize(make_float3(tintersection) / tintersection.w);
        attrib.normal = normalize(make_float3(sphere.trans.inverse().transpose() * make_float4(attrib.normal, 0)));
        attrib.mv = sphere.mv;
        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];
    float x, y, z;
    x = length(make_float3(sphere.trans.getRow(0)));
    y = length(make_float3(sphere.trans.getRow(1)));
    z = length(make_float3(sphere.trans.getRow(2)));
    result[0] = sphere.trans[3] - x;
    result[1] = sphere.trans[7] - y;
    result[2] = sphere.trans[11] - z;
    result[3] = sphere.trans[3] + x;
    result[4] = sphere.trans[7] + y;
    result[5] = sphere.trans[11] + z;
}