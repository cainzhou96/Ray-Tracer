#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"
#include "Geometries.h"

using namespace optix;

rtDeclareVariable(LightPayload, payload, rtPayload, );
rtDeclareVariable(float3, backgroundColor, , );

rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    payload.hit = 1; 
    payload.intersection = attrib.intersection; 
    float3 n = normalize(attrib.normal); 
    float3 w = normalize(attrib.wo); 

    if (dot(n, w) > 0) {
        payload.emission = attrib.mv.emission; 
    }
    else {
        payload.emission = backgroundColor; 
    }
}